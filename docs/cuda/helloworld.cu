//  First programs in CUDA.
//  Julien VALENTIN (2022)  julien.vlnt@gmail.com
//  "Hello, World !" program.
//  From Visual Studio Command Prompt : `nvcc helloCUDA.cu`
#include <hip/hip_runtime.h>

#include <stdio.h>

/*
    A function that is called by the host to run on the device. Here, it does
    not do anything. The host (CPU) will call this function to run on the 
    device (GPU).
*/
__global__ void do_nothing()
{
}

int main()
{
    // The host calls the function, distribute it in 1 block (memory splitting)
    // and 1 thread (number of cores). 
    do_nothing<<<1,1>>>();

    // Finally say Hello ! LOL !
    printf("Hello, World ! I am your C.P.U !");

    return 0;
}
